
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>


__device__ long long int mod(int base, int exponent, int den)
{
	unsigned int a = (  base % den   ) * ( base % den);
	// unsigned int a = base * base;
	unsigned long long int ret = 1;
	float size = (float) exponent / 2;
	if (  exponent == 0 )
	{
		return base % den;
	}
	else
	{
		while (1)
		{
			if ( size > 0.5  )
			{
				ret = ( ret * a  ) % den; size = size - 1.0;
			}
			else if (  size == 0.5  )
			{
				ret = (  ret * (  base % den  )   ) % den;
				// ret = ( ret  * base   ) % den;
				break;
			}
			else
			{
				break;

			}
		}
		return ret;
	}
}

__global__ void parallel_reduction(int *array, int *output, int mod)
{
	//extern __shared__ int sdata[];
	int tid = threadIdx.x;
	//int i = blockIdx.x * (blockDim.x) + tid;
	//sdata[tid] = array[i] ;
	//__syncthreads();
	
	
	//printf("%d;%d\n", array[tid], tid);
	for ( unsigned int s = 1; s < blockDim.x; s *= 2 ){
		if ( tid % ( 2 * s  ) == 0  ){
			if ( tid + s < blockDim.x ){
				array[tid] = ( (array[tid]  % mod  ) * (array[ tid + s ] % mod ) )% mod;

			}
		}
		__syncthreads();
		//printf("%d;%d\n", array[tid], tid);
	}

	/*
	for (int s = blockDim.x/2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] = (sdata[tid] * sdata[tid + s]) % mod;
		}
		__syncthreads();
	}
	*/
	if (tid == 0){
		//printf(  "%d\n", array[0]  );
		output[blockIdx.x] = array[0];
	}
}

__global__ void init_reduction(int value, int *array, int n)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= n) return;
	array[index] = value;
}


__global__ void rsa( int *num, int *key, int *den, unsigned int *result)
{
	int i = threadIdx.x;
	int temp;
	if (i == 0)
	{
		temp = mod( num[i], *key, *den);
		atomicExch( &result[i], temp );
	}

			
	
}

int main(){
	int nsize = 5;
	int num[5] = {104,101, 108, 108, 111};
	int key = 6558;
	int size = key / 2;
	int den = 91 * 97;
	int *d_num, *d_key, *d_den;
	unsigned int *d_res;
	unsigned int res[5] = {1,1,1,1,1};
	
	int num_blocks = (key + 2047 )/ 2048 ;
	int num_threads = 0;
	size <= 1024 ? num_threads = size:num_threads=1024;

	hipMalloc( (void **)&d_num, nsize * sizeof(int) );
	hipMalloc( (void **)&d_key, sizeof(int) );
	hipMalloc( (void **)&d_den, sizeof(int)  );
	hipMalloc( (void **)&d_res, nsize * sizeof(unsigned int) );
	
	hipMemcpy( d_num, &num, nsize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( d_key, &key, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( d_den, &den, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( d_res, res, nsize* sizeof(unsigned int), hipMemcpyHostToDevice);
	
	hipEvent_t start_p, stop_p;
    	float time;
    	hipEventCreate(&start_p);
    	hipEventCreate(&stop_p);
    	hipEventRecord(start_p, 0);
	
	rsa<<<1,5>>>(d_num, d_key, d_den, d_res);
	
	hipEventRecord(stop_p, 0);
	hipEventSynchronize(stop_p);
	hipEventElapsedTime(&time, start_p, stop_p);

	hipEventDestroy (start_p);
	hipEventDestroy (stop_p);
	
	hipMemcpy(res, d_res, nsize * sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	printf("Paper Time :   %f\n" , time );

	int base = (num[0] * num[0]) % den;
	int *input;
	int *output;
	int *ans;
	
	ans = (int *) malloc( size * sizeof(int)  );
	hipMalloc( (void **)&input, size * sizeof(int)  );
	hipMalloc( (void **)&output, size * sizeof(int)  );

	float new_time;
    	hipEventCreate(&start_p);
    	hipEventCreate(&stop_p);
    	hipEventRecord(start_p, 0);

	init_reduction<<<num_blocks,num_threads>>>(base, input, size );

	parallel_reduction<<<num_blocks,num_threads>>>(input, output, den);
	
	hipEventRecord(stop_p, 0);
	hipEventSynchronize(stop_p);
	hipEventElapsedTime(&new_time, start_p, stop_p);

	printf("Our Time : %f\n", new_time);
	printf("Speedup : %f\n", time/new_time);
	
	hipEventDestroy (start_p);
	hipEventDestroy (stop_p);

	hipMemcpy( ans, input, size * sizeof(int), hipMemcpyDeviceToHost);
	
 
	printf("%d - %d\n", ans[0], res[0] );
	hipFree(d_num);
	hipFree(d_key);
	hipFree(d_den);
	hipFree(d_res);
	hipFree(input);
	hipFree(output);
	free(ans);
	return 0;
}
