
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>


__device__ long long int mod(int base, int exponent, int den)
{
	unsigned int a = (  base % den   ) * ( base % den);
	// unsigned int a = base * base;
	unsigned long long int ret = 1;
	float size = (float) exponent / 2;
	if (  exponent == 0 ) {
		return base % den;
	}
	else
	{
		while (1)
		{
			if ( size > 0.5  )
			{
				ret = ( ret * a  ) % den; size = size - 1.0;
			}
			else if (  size == 0.5  )
			{
				ret = (  ret * (  base % den  )   ) % den;
				// ret = ( ret  * base   ) % den;
				break;
			}
			else
			{
				break;

			}
		}
		return ret;
	}
}

__global__ void parallel_reduction(int *array, int *output, int mod)
{
	extern __shared__ int sdata[];
	int tid = threadIdx.x;
	int i = blockIdx.x * (blockDim.x) + tid;
	sdata[tid] = array[i] ;
	__syncthreads();
	
	
	//SINGLE BLOCK SLOW
	///*
	for ( unsigned int s = 1; s < blockDim.x; s *= 2 ){
		if ( tid % ( 2 * s  ) == 0  ){
			if (tid + s < blockDim.x){
				sdata[tid] = ( (sdata[tid]  % mod  ) * (sdata[ tid + s ] % mod ) )% mod;
			}
		}
		__syncthreads();
	}
	//*/
	
	//SINGLE BLOCK MEDIUM
	/*
	for (int s = 1; s < blockDim.x ; s *= 2){
		int index = 2 * s * tid;
		if ( index  + s< blockDim.x  ){
			sdata[index] = ( (sdata[index]  % mod  ) * (sdata[ index + s ] % mod ) )% mod;

		}
		__syncthreads();

	}
	*/

	if (tid == 0){
		output[blockIdx.x] = sdata[0];
	}
}

__global__ void sumCommMultiBlock(const int *gArr, int arraySize, int *gOut, int mod, int blockSize) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    int sum = 1;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum = ( ( sum % mod  ) *  (gArr[i] % mod ) ) % mod ;
    __shared__ int shArr[1024];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
		shArr[thIdx] = ( (shArr[thIdx]  % mod  ) * (shArr[ thIdx + size ] % mod ) )% mod;
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}

__global__ void init_reduction(int value, int *array, int n)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= n) return;
	array[index] = value;
}


__global__ void rsa( int *num, int *key, int *den, unsigned int *result)
{
	int i = threadIdx.x;
	int temp;
	if (i == 0)
	{
		temp = mod( num[i], *key, *den);
		atomicExch( &result[i], temp );
	}
}

int main(){
	int nsize = 5;
	int num[5] = {104,101, 108, 108, 111};
	int key = 4000;
	int size = key / 2;
	int den = 91 * 97;
	int *d_num, *d_key, *d_den;
	unsigned int *d_res;
	unsigned int res[5] = {1,1,1,1,1};
	
	int num_blocks = (key + 2047 )/ 2048 ;
	int num_threads = 0;
	size <= 1024 ? num_threads = size:num_threads=1024;

	hipMalloc( (void **)&d_num, nsize * sizeof(int) );
	hipMalloc( (void **)&d_key, sizeof(int) );
	hipMalloc( (void **)&d_den, sizeof(int)  );
	hipMalloc( (void **)&d_res, nsize * sizeof(unsigned int) );
	
	hipMemcpy( d_num, &num, nsize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( d_key, &key, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( d_den, &den, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( d_res, res, nsize* sizeof(unsigned int), hipMemcpyHostToDevice);
	
	hipEvent_t start_p, stop_p;
    	float time;
    	hipEventCreate(&start_p);
    	hipEventCreate(&stop_p);
    	hipEventRecord(start_p, 0);
	
	rsa<<<1,5>>>(d_num, d_key, d_den, d_res);
	
	hipEventRecord(stop_p, 0);
	hipEventSynchronize(stop_p);
	hipEventElapsedTime(&time, start_p, stop_p);

	hipEventDestroy (start_p);
	hipEventDestroy (stop_p);
	
	hipMemcpy(res, d_res, nsize * sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	printf("Paper Time :   %f\n" , time );

	int base = (num[0] * num[0]) % den;
	int *input;
	int *output;
	int *ans;
	
	ans = (int *) malloc( size * sizeof(int)  );
	hipMalloc( (void **)&input, size * sizeof(int)  );
	hipMalloc( (void **)&output, size * sizeof(int)  );

	float new_time;
    	hipEventCreate(&start_p);
    	hipEventCreate(&stop_p);
    	hipEventRecord(start_p, 0);

	init_reduction<<<num_blocks,num_threads>>>(base, input, size );

	parallel_reduction<<<num_blocks,num_threads,size * sizeof(int)>>>(input, output, den);
	//sumCommMultiBlock<<<num_blocks,num_threads>>>(input,size,output,den,num_blocks);
	
	hipEventRecord(stop_p, 0);
	hipEventSynchronize(stop_p);
	hipEventElapsedTime(&new_time, start_p, stop_p);

	printf("Our Time : %f\n", new_time);
	printf("Speedup : %f\n", time/new_time);
	
	hipEventDestroy (start_p);
	hipEventDestroy (stop_p);
	
	hipDeviceSynchronize();

	hipMemcpy( ans, output, size * sizeof(int), hipMemcpyDeviceToHost);
	
	int final_ans = ans[0];
	for (int i = 1;i < num_blocks;i++){
		final_ans = ( (final_ans % den) * (ans[i] % den ) ) % den;
	}
	
	printf("%d -  %d\n", final_ans, res[0] );
	hipFree(d_num);
	hipFree(d_key);
	hipFree(d_den);
	hipFree(d_res);
	hipFree(input);
	hipFree(output);
	free(ans);
	return 0;
}
