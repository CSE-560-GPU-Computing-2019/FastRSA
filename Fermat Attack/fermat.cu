#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
__global__ void
single_mult(const int n, const char* num, int len, char *result, const int index)
{
    int carry = 0;
    
    for (int i = len - 1; i >= 0; --i)
    {
        int m = num[i] - '0';
        result[i] = (m * n + carry) % 10;
        carry = m/10;
    }
}

__global__ void
big_multiply(const char *num1, const char* num2, char *result, int len1, int len2)
{
    
}


__device__ __host__ uint64_t
fast_sqroot(uint64_t n)
{

    uint64_t ret = 0;
    for (uint64_t h = n/2, l = 1, m = (h+l)/2; ; )
    {
        //printf("%lu %lu %lu %lu %lu\n", h, l, m, m*m, n);
        if (m*m == n)
        {
            ret = m;
            break;
        }
        else if (m*m < n)
        {
            l = m + 1;    
        }
        else
        {
            h = m - 1;
        }

        m = (h+l)/2;
        
        if (h <= l)
        {
            ret = m;
            break;
        }

        
    }
    
    return ret;
}

__global__ void
big_add(const char *a, const char *b, char *c)
{
    
}
*/
__device__ int
is_square(uint64_t t)
{
    return (uint64_t)sqrtf(t) * (uint64_t)sqrtf(t) == t;
}

__global__ void
fermat_factorization(const uint64_t n, uint64_t *p, uint64_t *q, const uint64_t a, int *flag)
{
    //if (*flag > 0) return;
    uint64_t offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset + a >= n)
        return;

    if (n % 2 == 0)
    {
        *p = 2;
        *q = n>>1;
        return;
    }    
    
    uint64_t t = (a + offset)*(a + offset) - n;
    if (is_square(t))
    {
        //printf("t: %lu\n", t);
        uint64_t b = (uint64_t)sqrtf(t);
        //printf("b: %lu, a: %lu, a-b: %lu, a+b: %lu\n", b, a, (a-b), (a+b));
        //printf("p*q: %lu, n: %lu\n", ((a-b) * (a+b)), n);
        uint64_t pr = (a+offset-b) * (a+offset+b);
        if (pr == n && a+offset>b)
        {
            
            *p = a+offset-b;
            *q = a+offset+b;
            //*flag += 1;
            printf("p: %lu, q: %lu\n", *p, *q);
            asm("trap;");
        }  
    }

}


__global__ void
big_product(unsigned int *a, unsigned int *b, unsigned int *accumulator, int n)
{
    int multiplier = 0;
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= n) return;
    unsigned int multiplicand = index;
    unsigned int product = 0;

    while(multiplier < n) 
    {
        product = a[multiplier] * b[multiplicand];
        atomicAdd(&accumulator[multiplier + index], product<<24>>24);
        atomicAdd(&accumulator[multiplier + index + 1], product>>8);
        multiplier++;
    }
	return;
}

__global__ void 
long_bytes(const unsigned int *input, char *output, int length)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bd = blockDim.x;
    int index = bx*bd + tx;
    if (index >= length) return;
    output[index] = (char)input[index];
}

__global__ void 
bytes_long(const char *input, unsigned int *output, const int length)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bd = blockDim.x;
    int index = bx*bd + tx;
    if (index >= length) return;
    output[index] = (uint64_t)input[index];
}


__host__ int
main()
{
    /*
    char *message = (char *)malloc(sizeof(char) * 512);
    fgets(message, 512, stdin);
    message[strlen(message) - 1] = '\0';
    int message_size = strlen(message);
    unsigned int *plaintext = (unsigned int *)malloc(sizeof(unsigned int) * message_size);
    unsigned int *plaintext_device;
    char *message_device;
    cudaMalloc((void **)&plaintext_device, sizeof(unsigned int) * message_size);
    cudaMalloc((void **)&message_device, message_size);
    cudaMemcpy(message_device, message, message_size, cudaMemcpyHostToDevice);

    bytes_long <<<(message_size + 1023)/1024, 1024>>>(message_device, plaintext_device, message_size);

    cudaMemcpy(plaintext, plaintext_device, sizeof(unsigned int) * message_size, cudaMemcpyDeviceToHost);
    for (int i = 0 ; i < message_size ; ++i)
    {
        std::cout << plaintext[i];
    }
    std::cout<<"\n";
    
    int num_digits = 0;
    for (int i = 0 ; i < message_size ; ++i)
    {
        int cur = plaintext[i];
        while (cur > 0)
        {
            num_digits++;
            cur /= 10;
        }
    }
    */
    
    uint64_t n = 1103191240211;
    uint64_t a = (uint64_t)sqrt(n);
    printf("a: %lu\n", a);
    uint64_t *p, *q;
    int *flag;
    uint64_t *ph = (uint64_t *)malloc(sizeof(uint64_t));
    uint64_t *qh = (uint64_t *)malloc(sizeof(uint64_t));
    hipMalloc((void **)&p, sizeof(uint64_t));
    hipMalloc((void **)&q, sizeof(uint64_t));
    hipMalloc((void **)&flag, sizeof(int));
    float elapsed = 0;

    fermat_factorization<<<(n-a+1023)/1024, 1024>>>(n, p, q, a, flag);

    hipMemcpy(ph, p, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(qh, q, sizeof(uint64_t), hipMemcpyDeviceToHost);

    printf("%lu %lu\n", *ph, *qh);
    printf("Time elapsed in gpu %.2f ms\n", elapsed);
    
}