#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int
is_square(uint64_t t)
{
    return (uint64_t)sqrtf(t) * (uint64_t)sqrtf(t) == t;
}

__global__ void
fermat_factorization(const uint64_t n, uint64_t *p, uint64_t *q, const uint64_t a, int *flag)
{
    uint64_t offset = threadIdx.x + blockDim.x * blockIdx.x;

    if (offset + a >= n)
        return;

    if (n % 2 == 0)
    {
        *p = 2;
        *q = n>>1;
        return;
    }    
    
    uint64_t t = (a + offset)*(a + offset) - n;

    if (is_square(t))
    {
        uint64_t b = (uint64_t)sqrtf(t);
        uint64_t pr = (a+offset-b) * (a+offset+b);
        if (pr == n && a+offset>b)
        {
            
            *p = a+offset-b;
            *q = a+offset+b;
            
            //asm("trap;");
        }  
    }

}


__global__ void
big_product(unsigned int *a, unsigned int *b, unsigned int *accumulator, int n)
{
    int multiplier = 0;
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= n) return;
    unsigned int multiplicand = index;
    unsigned int product = 0;

    while(multiplier < n) 
    {
        product = a[multiplier] * b[multiplicand];
        atomicAdd(&accumulator[multiplier + index], product<<24>>24);
        atomicAdd(&accumulator[multiplier + index + 1], product>>8);
        multiplier++;
    }
	return;
}

__global__ void 
long_bytes(const unsigned int *input, char *output, int length)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bd = blockDim.x;
    int index = bx*bd + tx;
    if (index >= length) return;
    output[index] = (char)input[index];
}

__global__ void 
bytes_long(const char *input, unsigned int *output, const int length)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bd = blockDim.x;
    int index = bx*bd + tx;
    if (index >= length) return;
    output[index] = (uint64_t)input[index];
}


__host__ int
main()
{
    
    uint64_t n = 4335743309;//69417725381;//18512544;//4335743309;//1103191240211;//18512544
    uint64_t a = (uint64_t)sqrt(n);
    printf("a: %lu\n", a);
    uint64_t *p, *q;
    int *flag;
    uint64_t *ph = (uint64_t *)malloc(sizeof(uint64_t));
    uint64_t *qh = (uint64_t *)malloc(sizeof(uint64_t));
    hipMalloc((void **)&p, sizeof(uint64_t));
    hipMalloc((void **)&q, sizeof(uint64_t));
    hipMalloc((void **)&flag, sizeof(int));
   

   	float elapsed = 0;


    hipEvent_t d_start, d_stop;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);
	hipEventRecord(d_start, 0);
    fermat_factorization<<<(n-a+1023)/1024, 1024>>>(n, p, q, a, flag);
    hipEventRecord(d_stop);
	hipEventSynchronize(d_stop);
	hipEventElapsedTime(&elapsed, d_start, d_stop);
	hipEventDestroy(d_start);
	hipEventDestroy(d_stop);
    hipMemcpy(ph, p, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(qh, q, sizeof(uint64_t), hipMemcpyDeviceToHost);
    
    //printf("Time taken: %fms\n\n", runTime*1000);
    //printf("%lu %lu\n", *ph, *qh);
    printf("p: %lu, q: %lu\n", *ph, *qh);
    printf("Time elapsed in gpu %.4f ms\n", elapsed);
    
}
