#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ uint64_t
gcd(uint64_t u, uint64_t v)
{
    int shift;
    if (u == 0) return v;
    if (v == 0) return u;
    shift = __clzll(__brevll(u | v));
    u >>= __clzll(__brevll(u));
    do 
    {
        v >>= __clzll(__brevll(v));
        if (u > v) 
        {
            uint64_t t = v;
            v = u;
            u = t;
        }  
        v = v - u;
    } while (v != 0);
    return u << shift;
}


__global__ void
seive(uint64_t *arr, const uint64_t val, const uint64_t limit)
{
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int bd = blockDim.x;

	int idx = tx + bx * bd;

	if (idx * val > limit || idx == 0 || idx == 1)
		return;

	arr[idx * val] = 1;
}


__global__ void
pollard_factorization(const uint64_t n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < 2)
    {
    	return;
    }

    int b = idx;

    uint64_t *arr = (uint64_t *)malloc((b+1) * sizeof(uint64_t));
    memset(arr, 0, (b+1) * sizeof(uint64_t));
    uint64_t M = 1;
    for (int i = 2; i <= b; ++i)
    {
    	if (arr[i] == 0)
    	{
    		seive<<<(b+1023)/1024, 1024>>>(arr, i, b);
    		hipDeviceSynchronize();
    		M *= (uint64_t)ceil(pow(i, (uint64_t)(floor(__logf(b)/__logf(i)))));
    	}	
    }
    free(arr);
    uint64_t am = pow(2, M);
    am--;
    uint64_t p = gcd(am, n), q = n/p;
    if (p != 1 && p != n)
    {
    	printf("%llu %llu\n", p, q);
    }
}

__host__ int
main(int argc, char *argv[])
{
    
    uint64_t n = 299;//, b = atoi(argv[1]);
    float elapsed = 0;


    hipEvent_t d_start, d_stop;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);
	hipEventRecord(d_start, 0);
    pollard_factorization<<<40, 20>>>(n);
    hipEventRecord(d_stop);
	hipEventSynchronize(d_stop);
	hipEventElapsedTime(&elapsed, d_start, d_stop);
	hipEventDestroy(d_start);
	hipEventDestroy(d_stop);

    
    //printf("Time taken: %fms\n\n", runTime*1000);
    //printf("%lu %lu\n", *ph, *qh);
    printf("Time elapsed in gpu %.2f ms\n", elapsed);
    
}
